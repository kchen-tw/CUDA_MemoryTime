﻿#include <iostream>
#include <hip/hip_runtime.h>

int main(int argc, char* argv[]) {
    int N;

    // 檢查是否有提供命令列參數
    if (argc > 1) {
        // 將第一個命令列參數轉換為整數
        N = std::atoi(argv[1]);
    }
    else {
        // 如果沒有提供命令列參數，設定 N 為預設值 10
        N = 20;
    }

    // 定義陣列大小
    const int arraySize = 1 << N;
    std::cout << "Array Size = 2^" << N << std::endl;
    // 在主機上分配和初始化陣列
    int* h_data = new int[arraySize];
    for (int i = 0; i < arraySize; ++i) {
        h_data[i] = i + 1;
    }

    // 在裝置上分配陣列
    int* d_data;
    hipMalloc(&d_data, sizeof(int) * arraySize);

    // 定義CUDA事件
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // 記錄開始時間
    hipEventRecord(start);

    // 將資料從主機複製到裝置
    hipMemcpy(d_data, h_data, sizeof(int) * arraySize, hipMemcpyHostToDevice);

    // 記錄結束時間
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // 計算執行時間
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "hipMemcpy HostToDevice 執行時間: " << milliseconds << " 毫秒" << std::endl;

    // 記錄開始時間
    hipEventRecord(start);

    // 將資料從裝置複製回主機
    hipMemcpy(h_data, d_data, sizeof(int) * arraySize, hipMemcpyDeviceToHost);

    // 記錄結束時間
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // 計算執行時間
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "hipMemcpy DeviceToHost 執行時間: " << milliseconds << " 毫秒" << std::endl;

    // 釋放記憶體
    delete[] h_data;
    hipFree(d_data);

    // 銷毀CUDA事件
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
